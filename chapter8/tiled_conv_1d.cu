#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

// CUDA API error handling macro
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define MAX_MASK_WIDTH 5
#define BLOCK_WIDTH 32
#define TILE_WIDTH 32

__constant__ float M[MAX_MASK_WIDTH];

__global__ void tiled_conv_1d_basic_k(float* N, float* P, int Mask_Width, int Width)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ float Nds[TILE_WIDTH + MAX_MASK_WIDTH - 1];

  int n = Mask_Width / 2;

  int halo_idx_left = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
  if (threadIdx.x >= blockDim.x - n) {
    Nds[threadIdx.x - (blockDim.x - n)] = (halo_idx_left < 0 ) ? 0 : N[halo_idx_left];
  }

  Nds[n + threadIdx.x] = N[blockIdx.x * blockDim.x + threadIdx.x];

  // TODO: There might be an error in the book.
  // Need to check this to avoid overriding Nds[n + threadIdx]
  // when threadIdx.x = blockIdx.x = 0;
  bool is_Nds_loaded = Nds[n + blockIdx.x + threadIdx.x] != NULL;

  int halo_idx_right = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
  if (threadIdx.x < n && !is_Nds_loaded) {
    Nds[n + blockIdx.x + threadIdx.x] = (halo_idx_right >= Width) ? 0 : N[halo_idx_right];
  }

  __syncthreads();

  float Pvalue = 0;
  for (int j = 0; j < Mask_Width; j++) {
    Pvalue += M[j] * Nds[threadIdx.x + j];
  }

  P[i] = Pvalue;
}

float* iota(int m, int n)
{
  int len = m * n;
  float* mat = new float[len];

  for (int j = 0; j < n; j++) {
    for (int i = 0; i < m; i++) {
      mat[i * n + j] = i * n + j;
    }
  }

  return mat;
}

int main() {
  int width = 32;

  float* N = iota(1, width);
  float *d_P, *d_N;

  float *P = new float[width];
  float h_M[MAX_MASK_WIDTH];

  h_M[0] = 3;
  h_M[1] = 7;
  h_M[2] = 9;
  h_M[3] = 5;
  h_M[4] = 1;

  gpuErrchk(hipMalloc((void **)&d_N, width * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&d_P, width * sizeof(float)));
  gpuErrchk(hipMemcpy(d_N, N, width * sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(M), h_M, MAX_MASK_WIDTH * sizeof(float)));

  int numBlocks = width / BLOCK_WIDTH;

  if (width % BLOCK_WIDTH) numBlocks++;

  tiled_conv_1d_basic_k<<<numBlocks, BLOCK_WIDTH>>>(d_N, d_P, MAX_MASK_WIDTH, width);

  gpuErrchk(hipMemcpy(P, d_P, width * sizeof(float), hipMemcpyDeviceToHost));

  for (int k = 0; k < width; k++) {
    if ((k + 1) % width == 0) {
      cout << P[k] << endl;
    } else {
      cout << P[k] << " ";
    }
  }

  gpuErrchk(hipFree(d_N));
  gpuErrchk(hipFree(d_P));

  delete P;
  delete N;
}