#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

// CUDA API error handling macro
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define MAX_MASK_WIDTH 5
#define TILE_WIDTH 4

__constant__ float M[MAX_MASK_WIDTH];

__global__ void tiled_conv_1d_basic_k(float* N, float* P, int Mask_Width, int Width)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ float Nds[TILE_WIDTH + MAX_MASK_WIDTH - 1];

  int n = Mask_Width / 2;

  int halo_idx_left = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
  if (threadIdx.x >= blockDim.x - n) {
    Nds[threadIdx.x - (blockDim.x - n)] = (halo_idx_left < 0 ) ? 0 : N[halo_idx_left];
  }

  Nds[n + threadIdx.x] = N[blockIdx.x * blockDim.x + threadIdx.x];

  int halo_idx_right = (blockIdx.x + 1)*blockDim.x + threadIdx.x;
  if (threadIdx.x < n) {
    Nds[n + blockIdx.x + threadIdx.x] = (halo_idx_right >= Width) ? 0 : N[halo_idx_right];
  }

  __syncthreads();

  float Pvalue = 0;
  for (int j = 0; j < Mask_Width; j++) {
    Pvalue += M[j] * Nds[threadIdx.x + j];
  }

  P[i] = Pvalue;
}

float* iota(int m, int n)
{
  int len = m * n;
  float* mat = new float[len];

  for (int j = 0; j < n; j++) {
    for (int i = 0; i < m; i++) {
      mat[i * n + j] = i * n + j;
    }
  }

  return mat;
}

int main() {
  int width = 16;

  float* N = iota(1, width);
  float *d_P, *d_N;

  float *P = new float[width];
  float h_M[MAX_MASK_WIDTH];

  h_M[0] = 3;
  h_M[1] = 7;
  h_M[2] = 9;
  h_M[3] = 5;
  h_M[4] = 1;

  gpuErrchk(hipMalloc((void **)&d_N, width * sizeof(float)));
  gpuErrchk(hipMalloc((void **)&d_P, width * sizeof(float)));
  gpuErrchk(hipMemcpy(d_N, N, width * sizeof(float), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(M), h_M, MAX_MASK_WIDTH * sizeof(float)));

  tiled_conv_1d_basic_k<<<4, 4>>>(d_N, d_P, MAX_MASK_WIDTH, width);

  gpuErrchk(hipMemcpy(P, d_P, width * sizeof(float), hipMemcpyDeviceToHost));

  for (int k = 0; k < width; k++) {
    if ((k + 1) % width == 0) {
      cout << P[k] << endl;
    } else {
      cout << P[k] << " ";
    }
  }

  gpuErrchk(hipFree(d_N));
  gpuErrchk(hipFree(d_P));

  delete P;
  delete N;
}